#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#include ""
#include "CImg.h"
#include <stdio.h>
#include <iostream>

using namespace std;
using namespace cimg_library;

void computeParallelFlow(CImg<unsigned int> *firstFrame, CImg<float> *Ix, CImg<float> *Iy, CImg<float> *It, CImg<float> *vOdd, CImg<float> *vEven, CImg<float> *uOdd, CImg<float> *uEven);
void computeParallelDerivatives(CImg<unsigned int> *firstFrame, CImg<unsigned int> *secondFrame, CImg<float> *Ix, CImg<float> *Iy, CImg<float> *It);

//Kernel function to calculate partial derivatives.
__global__ void ppDerivatives(unsigned int* frame1, unsigned int* frame2, float* I_dx, float* I_dy, float* I_dt)
{
	int threadY = blockDim.y * blockIdx.y + threadIdx.y;
	int threadX = blockDim.x * blockIdx.x + threadIdx.x;

	if (!((threadY + 1) >= 600 && !((threadX + 1) >= 600))) {

		I_dx[600 * threadY + threadX] = (0.25) * ((frame1)[600 * (threadY + 1) + threadX] - (frame1)[600 * threadY + threadX] + (frame1)[600 * (threadY + 1) + (threadX + 1)] - (frame1)[600 * (threadY) + (threadX + 1)]
			+ (frame2)[600 * (threadY + 1) + (threadX)] - (frame2)[600 * (threadY) + (threadX)] + (frame2)[600 * (threadY + 1) + (threadX + 1)] - (frame2)[600 * (threadY) + (threadX + 1)]);

		I_dy[600 * threadY + threadX] = (0.25) * ((frame1)[600 * (threadY) + (threadX + 1)] - (frame1)[600 * threadY + threadX] + (frame1)[600 * (threadY + 1) + (threadX + 1)] - (frame1)[600 * (threadY + 1) + threadX]
			+ (frame2)[600 * (threadY) + (threadX + 1)] - (frame2)[600 * (threadY) + (threadX)] + (frame2)[600 * (threadY + 1) + (threadX + 1)] - (frame2)[600 * (threadY + 1) + (threadX)]);

		I_dt[600 * threadY + threadX] = (0.25) * ((frame2)[600 * (threadY) + (threadX)] - (frame1)[600 * threadY + threadX] + (frame2)[600 * (threadY) + (threadX + 1)] - (frame1)[600 * (threadY) + (threadX + 1)]
			+ (frame2)[600 * (threadY + 1) + (threadX)] - (frame1)[600 * (threadY + 1) + threadX] + (frame2)[600 * (threadY + 1) + (threadX + 1)] - (frame1)[600 * (threadY + 1) + (threadX + 1)]);
	}
	else {
		printf("Highter than 600.");
	}

}

//Device-side functions to calculate averages and alpha
__device__ float calculateAlpha(float dI, float dx, float dy, float dt, float uAverage, float vAverage) {
	
	float weight = 1.2;

	float numerator = dI * (dx * uAverage + dy * vAverage + dt);

	float denominator = (1 + weight * ((powf(dx, 2.0f) + powf(dy, 2.0f))));

	return numerator / denominator;
}

__device__ float calculateLocalAverages(float* flowVector, int threadX, int threadY) {

	float first = 0;
	float second = 0;

	if (!((threadY + 1) >= 600 && !((threadX + 1) >= 600))) {
		first = ((flowVector)[600 * (threadY)+(threadX - 1)] + (flowVector)[600 * (threadY + 1) + (threadX)] + (flowVector)[600 * (threadY)+(threadX + 1)] + (flowVector)[600 * (threadY - 1) + (threadX)]);
		second = ((flowVector)[600 * (threadY - 1) + (threadX - 1)] + (flowVector)[600 * (threadY + 1) + (threadX - 1)] + (flowVector)[600 * (threadY + 1) + (threadX + 1)] + (flowVector)[600 * (threadY - 1) + (threadX + 1)]);
	}

	return (1/6) * first + (1/12) * second;

}

//Kernel function to calculate optical flow
__global__ void ppFlow(float* I_dx, float* I_dy, float* I_dt, float* pu_Odd, float* pv_Odd, float* pu_Even, float* pv_Even)
{
	int threadY = blockDim.y * blockIdx.y + threadIdx.y;
	int threadX = blockDim.x * blockIdx.x + threadIdx.x;

	float alpha = 0;
	float beta = 0;
	
	//Calculate the approximations of the Laplacians u and v.
	float uLocalAverage = calculateLocalAverages(pu_Odd, threadX, threadY);
	float vLocalAverage = calculateLocalAverages(pv_Odd, threadX, threadY);

	//Calulate the "alpha" term
	alpha = calculateAlpha(I_dx[600 * (threadY)+(threadX)], I_dx[600 * (threadY)+(threadX)], I_dy[600 * (threadY)+(threadX)], I_dt[600 * (threadY)+(threadX)], uLocalAverage, vLocalAverage);
	beta = calculateAlpha(I_dy[600 * (threadY)+(threadX)], I_dx[600 * (threadY)+(threadX)], I_dy[600 * (threadY)+(threadX)], I_dt[600 * (threadY)+(threadX)], uLocalAverage, vLocalAverage);

	//Calculate the next u from the previous u
	(pu_Even)[600 * (threadY) + (threadX)] = uLocalAverage - 0.8 * alpha;

	//Calculate the next v from the previous v
	(pv_Even)[600 * (threadY) + (threadX)] = vLocalAverage - 0.8 * beta;
}

int main()
{

	//======================READ IMAGES=====================
	CImg<unsigned int> frame1;
	CImg<unsigned int> frame2;

	try {
		frame1.load_bmp("S:\\Projects\\CUDA\\CSC 592 - Final Project\\CSC 592 - Final Project\\Testing Images\\image1RGB.bmp");
		frame2.load_bmp("S:\\Projects\\CUDA\\CSC 592 - Final Project\\CSC 592 - Final Project\\Testing Images\\image2RGB.bmp");
	}
	catch (CImgIOException) {
		std::cout << "Unable to find specified images, quiting execution." << std::endl;
		exit(EXIT_FAILURE);
	}
	//======================================================

	//==================COMPARE IMAGE SIZES=================
	if (!((frame1.width() == frame2.width()) && (frame1.height() == frame2.height()))) {
		std::cout << "The specified images are not the same size, quiting execution." << std::endl;
		exit(EXIT_FAILURE);
	} 
	//======================================================

	CImg<float> I_dx(frame1.width(), frame1.height());
	CImg<float> I_dy(frame1.width(), frame1.height());
	CImg<float> I_dt(frame1.width(), frame1.height());

	CImg<float> u_Even(frame1.width(), frame1.height());
	CImg<float> v_Even(frame1.width(), frame1.height());
	CImg<float> u_Odd(frame1.width(), frame1.height());
	CImg<float> v_Odd(frame1.width(), frame1.height());

	int T = 100;
	int n = 0;

	while (n <= T) {

		computeParallelFlow(&frame1, &I_dx, &I_dy, &I_dt, &v_Odd, &v_Even, &u_Odd, &u_Even);

		//Swap pointers
		{
			v_Odd.swap(v_Even);
			u_Odd.swap(u_Even);
		}

		//Increment n
		n += 1;
	}


    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
	hipError_t cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

	//===================PLOT THE VECTORS===================

	CImg<float> needleMap(frame1.width(), frame1.height());

	needleMap.fill(255);

	int black[] = { 0,0,0 };

	CImg<> uAverage(u_Odd.width(), u_Odd.height());

	uAverage = (u_Odd + u_Even) / 2;

	CImg<> vAverage(v_Odd.width(), v_Odd.height());

	vAverage = (v_Odd + v_Even) / 2;

	for (int i = 0; i < needleMap.height(); i++) {
		for (int j = 0; j < needleMap.width(); j++) {
			if (j % 4 == 0 && i % 8 == 0) {
				needleMap.draw_line(j, i, j + (uAverage(j, i)), i + (vAverage(j, i)), black);
			}
		}
	}

	needleMap.display();

	needleMap.save_bmp("C:\\Users\\Robert\\Desktop\\HSFlow.bmp");
	//======================================================
    return 0;
}
//Helper function to compute the optical flow
void computeParallelFlow(CImg<unsigned int> *firstFrame, CImg<float> *Ix, CImg<float> *Iy, CImg<float> *It, CImg<float> *vOdd, CImg<float> *vEven, CImg<float> *uOdd, CImg<float> *uEven) {
	float* d_Ix;
	float* d_Iy;
	float* d_It;

	float* d_v_Odd;
	float* d_v_Even;
	float* d_u_Odd;
	float* d_u_Even;

	size_t size = (*firstFrame).height() *(*firstFrame).width() * sizeof(float);

	//Allocate and copy device side memory
	hipMalloc(&d_Ix, size);
	hipMemcpy(d_Ix, (*Ix).data(), size, hipMemcpyHostToDevice);

	hipMalloc(&d_Iy, size);
	hipMemcpy(d_Iy, (*Iy).data(), size, hipMemcpyHostToDevice);

	hipMalloc(&d_It, size);
	hipMemcpy(d_It, (*It).data(), size, hipMemcpyHostToDevice);

	hipMalloc(&d_v_Odd, size);
	hipMemcpy(d_v_Odd, (*vOdd).data(), size, hipMemcpyHostToDevice);

	hipMalloc(&d_v_Even, size);
	hipMemcpy(d_v_Even, (*vEven).data(), size, hipMemcpyHostToDevice);

	hipMalloc(&d_u_Odd, size);
	hipMemcpy(d_u_Odd, (*uOdd).data(), size, hipMemcpyHostToDevice);

	hipMalloc(&d_u_Even, size);
	hipMemcpy(d_u_Even, (*uEven).data(), size, hipMemcpyHostToDevice);
	
	hipError_t err = hipGetLastError();
	if (err != hipSuccess)
		printf("Error: %s\n", hipGetErrorString(err));

	//Create a kernel to do work on the GPU
	dim3 numBlocks = { 600,600,1 };
	dim3 numThreads = { 1,1,1 };

	cout << "Calling Kernel..." << endl;

	ppFlow << < numBlocks, numThreads >> > (d_Ix, d_Iy, d_It, d_u_Odd, d_v_Odd, d_u_Even, d_v_Even);

	err = hipGetLastError();
	if (err != hipSuccess)
		printf("Error: %s\n", hipGetErrorString(err));

	hipMemcpy((*uOdd).data(), d_u_Odd, size, hipMemcpyDeviceToHost);
	hipMemcpy((*vOdd).data(), d_v_Odd, size, hipMemcpyDeviceToHost);
	hipMemcpy((*uEven).data(), d_u_Even, size, hipMemcpyDeviceToHost);
	hipMemcpy((*vEven).data(), d_v_Even, size, hipMemcpyDeviceToHost);

	 err = hipGetLastError();
	if (err != hipSuccess)
		printf("Error: %s\n", hipGetErrorString(err));


	hipFree(d_Ix);
	hipFree(d_Iy);
	hipFree(d_It);
	hipFree(d_u_Odd);
	hipFree(d_v_Odd);
	hipFree(d_u_Even);
	hipFree(d_v_Even);

}
//Helper function to compute partial derivatives in parallel
void computeParallelDerivatives(CImg<unsigned int> *firstFrame, CImg<unsigned int> *secondFrame, CImg<float> *Ix, CImg<float> *Iy, CImg<float> *It)
{

	unsigned int* d_frame1;
	unsigned int* d_frame2;
	float* d_Ix;
	float* d_Iy;
	float* d_It;

	size_t size = (*firstFrame).height() *(*firstFrame).width() * sizeof(unsigned int);

	hipMalloc(&d_frame1, size);
	hipMemcpy(d_frame1, (*firstFrame).data(), size, hipMemcpyHostToDevice);

	hipMalloc(&d_frame2, size);
	hipMemcpy(d_frame2, (*secondFrame).data(), size, hipMemcpyHostToDevice);

	size = (*firstFrame).height() *(*firstFrame).width() * sizeof(float);

	hipMalloc(&d_Ix, size);
	hipMalloc(&d_Iy, size);
	hipMalloc(&d_It, size);

	dim3 numBlocks = { 599,599,1 };
	dim3 numThreads = { 1,1,1 };

	cout << "Calling Kernel..." << endl;

	ppDerivatives <<< numBlocks, numThreads >>> (d_frame1, d_frame2, d_Ix, d_Iy, d_It);
	
	hipError_t err = hipGetLastError();
	if (err != hipSuccess)
		printf("Error: %s\n", hipGetErrorString(err));

	hipMemcpy((*Ix).data(), d_Ix, size, hipMemcpyDeviceToHost);
	hipMemcpy((*Iy).data(), d_Iy, size, hipMemcpyDeviceToHost);
	hipMemcpy((*It).data(), d_It, size, hipMemcpyDeviceToHost);

	hipFree(d_frame1);
	hipFree(d_frame2);
	hipFree(d_Ix);
	hipFree(d_Iy);
	hipFree(d_It);

}